#include "hip/hip_runtime.h"
#include "util/TextureUtils.h"
#include "util/DebugUtils.h"
#include "util/Timer.h"

#define STB_IMAGE_IMPLEMENTATION
#include "ext/stb/stb_image.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "ext/stb/stb_image_write.h"

#include "nvtt/nvtt_lowlevel.h"

#include <fstream>
#include <string>
#include <filesystem>
#include "TextureUtils.h"

namespace jazzfusion
{

    const std::vector<std::string> &TextureManager::GetTextureFiles()
    {
        static const std::vector<std::string> textureFiles = {
            "rocky_trail",
            "gray_rocks",
            "seaworn_stone_tiles",
            "beige_wall_001",
            "wood_planks",
            "bark_willow_02",
        };
        return textureFiles;
    }

    __global__ void fillFirstMipmapKernel(
        unsigned char *dMipmap,
        const unsigned char *dSource,
        int nInputBufferChannels,
        int nChannels,
        int currentSize)
    {
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i >= currentSize * currentSize)
            return;

        // Copy the existing channels
        for (int ch = 0; ch < nInputBufferChannels; ++ch)
        {
            dMipmap[i * nChannels + ch] = dSource[i * nInputBufferChannels + ch];
        }

        // Fill in leftover channels with 255
        for (int ch = nInputBufferChannels; ch < nChannels; ++ch)
        {
            dMipmap[i * nChannels + ch] = 255;
        }
    }

    __global__ void fillMipmapKernel(
        unsigned char *dDst,       // Current LOD
        const unsigned char *dSrc, // Previous (larger) LOD
        int nChannels,
        int dstSize) // = currentSize
    {
        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x >= dstSize || y >= dstSize)
            return;

        // We assume that srcSize = 2 * dstSize
        // Index for the downsampled LOD
        int dstIdx = y * dstSize + x;

        int srcSize = dstSize * 2;
        int i00 = (y * 2) * srcSize + (x * 2);
        int i01 = (y * 2) * srcSize + (x * 2 + 1);
        int i10 = (y * 2 + 1) * srcSize + (x * 2);
        int i11 = (y * 2 + 1) * srcSize + (x * 2 + 1);

        for (int ch = 0; ch < nChannels; ++ch)
        {
            float val =
                static_cast<float>(dSrc[i00 * nChannels + ch]) +
                static_cast<float>(dSrc[i01 * nChannels + ch]) +
                static_cast<float>(dSrc[i10 * nChannels + ch]) +
                static_cast<float>(dSrc[i11 * nChannels + ch]);
            val *= 0.25f; // Average
            if (val > 255.0f)
                val = 255.0f;

            dDst[dstIdx * nChannels + ch] = static_cast<unsigned char>(val);
        }
    }

    inline int log2i(int a)
    {
        assert(a != 0);
        int targetlevel = 0;
        while (a >>= 1)
            ++targetlevel;
        return targetlevel;
    }

    inline bool IsPowerOfTwo(int a)
    {
        return (a & (a - 1)) == 0;
    }

    void TextureManager::init()
    {
        std::filesystem::path cwd = std::filesystem::current_path();

        std::vector<std::string> filePaths;
        for (const auto &textureFile : GetTextureFiles())
        {
            filePaths.emplace_back("data/" + textureFile + "_albedo.png");
            filePaths.emplace_back("data/" + textureFile + "_normal.png");
            filePaths.emplace_back("data/" + textureFile + "_rough.png");
        }
        filePaths.emplace_back("data/water1.jpg");

        m_textures.resize(filePaths.size());

        if (!std::filesystem::exists("tex"))
        {
            std::filesystem::create_directory("tex");
        }

        for (int i = 0; i < filePaths.size(); ++i)
        {
            std::string filePath = filePaths[i];
            textureNameIdLookup[filePath] = i;

            // ScopeTimer timer("Generating texture for " + filePath);

            int fileNamePosStart = filePath.find('/') + 1;
            int fileNamePosEnd = filePath.find(".png");

            std::string cacheFileNameBase = "tex/" + filePath.substr(fileNamePosStart, fileNamePosEnd - fileNamePosStart);

            auto &texture = m_textures[i];
            auto &texObj = m_shaderTextures.texObjs[i];

            stbi_uc *buffer = stbi_load(filePath.c_str(), &texture.width, &texture.height, &texture.channel, 0);

            int nChannels = 4;
            int nInputBufferChannels = texture.channel;
            int bytesPerTile = 16;

            if (texture.channel == 3 || texture.channel == 4)
            {
                texture.format = hipCreateChannelDesc<cudaChannelFormatKindUnsignedBlockCompressed7>();
                nChannels = 4;
                bytesPerTile = 16;
            }
            else if (texture.channel == 2)
            {
                texture.format = hipCreateChannelDesc<cudaChannelFormatKindUnsignedBlockCompressed5>();
                nChannels = 2;
                bytesPerTile = 16;
            }
            else if (texture.channel == 1)
            {
                texture.format = hipCreateChannelDesc<cudaChannelFormatKindUnsignedBlockCompressed4>();
                nChannels = 1;
                bytesPerTile = 8;
            }
            else
            {
                assert(0);
            }

            assert(texture.width == texture.height);
            assert(IsPowerOfTwo(texture.width));

            int maxLod = log2i(texture.width) - 2;
            int numLods = maxLod + 1;

            CUDA_CHECK(hipMallocMipmappedArray(&texture.bufferArray, &texture.format, make_hipExtent(texture.width, texture.height, 0), numLods, 0));

            texture.resourceDesc = hipResourceDesc{};
            texture.resourceDesc.resType = hipResourceTypeMipmappedArray;
            texture.resourceDesc.res.mipmap.mipmap = texture.bufferArray;

            texture.texDesc = hipTextureDesc{};
            texture.texDesc.addressMode[0] = hipAddressModeWrap;
            texture.texDesc.addressMode[1] = hipAddressModeWrap;
            texture.texDesc.addressMode[2] = hipAddressModeWrap;
            texture.texDesc.borderColor[0] = 0.0f;
            texture.texDesc.borderColor[1] = 0.0f;
            texture.texDesc.borderColor[2] = 0.0f;
            texture.texDesc.borderColor[3] = 0.0f;
            texture.texDesc.disableTrilinearOptimization = 0;
            texture.texDesc.filterMode = hipFilterModeLinear;
            texture.texDesc.maxAnisotropy = 0;
            texture.texDesc.maxMipmapLevelClamp = (float)maxLod;
            texture.texDesc.minMipmapLevelClamp = 0.0f;
            texture.texDesc.mipmapFilterMode = hipFilterModeLinear;
            texture.texDesc.mipmapLevelBias = 0;
            texture.texDesc.normalizedCoords = 1;
            texture.texDesc.readMode = hipReadModeNormalizedFloat;
            texture.texDesc.seamlessCubemap = 0;
            texture.texDesc.sRGB = 0;

            CUDA_CHECK(hipCreateTextureObject(&texObj, &texture.resourceDesc, &texture.texDesc, nullptr));

            std::vector<std::vector<uint8_t>> mipmapBuffers(numLods);
            int currentSize = texture.width;

            for (int lod = 0; lod < numLods; ++lod)
            {
                // ScopeTimer timer("Generating mip level " + std::to_string(lod));

                hipArray_t currentMipLevelArray;
                CUDA_CHECK(hipGetMipmappedArrayLevel(&currentMipLevelArray, texture.bufferArray, lod));

                hipExtent currentMipLevelSize;
                CUDA_CHECK(hipArrayGetInfo(NULL, &currentMipLevelSize, NULL, currentMipLevelArray));

                assert(currentMipLevelSize.width == currentSize && currentMipLevelSize.height == currentSize);

                int pitch = currentSize / 4 * bytesPerTile;
                int height = currentSize / 4;

                std::string cacheFileName = cacheFileNameBase + "_lod_" + std::to_string(lod) + ".bin";
                std::string cacheFileNameDebug = cacheFileNameBase + "_lod_" + std::to_string(lod) + ".png";

                int expectedCompressedTextureSize = currentSize * currentSize / 16 * bytesPerTile;

                std::vector<std::byte> compressedImageBuffer(expectedCompressedTextureSize);

                if (std::filesystem::exists(cacheFileName))
                {
                    std::ifstream infile(cacheFileName, std::ifstream::in | std::ifstream::binary);
                    assert(infile.good());
                    infile.read(reinterpret_cast<char *>(compressedImageBuffer.data()), expectedCompressedTextureSize);
                    infile.close();
                }
                else
                {
                    mipmapBuffers[lod].resize(currentSize * currentSize * nChannels);

                    unsigned char *dSource = nullptr;
                    unsigned char *dMipmap = nullptr;

                    if (lod == 0)
                    // Example: create first LOD on GPU
                    {
                        size_t numPixels = currentSize * currentSize;
                        size_t sizeBytes = numPixels * nChannels * sizeof(unsigned char);

                        // Allocate device buffer for the source buffer (optional if you do it once)
                        CUDA_CHECK(hipMalloc(&dSource, numPixels * nInputBufferChannels * sizeof(unsigned char)));
                        // Copy from host
                        CUDA_CHECK(hipMemcpy(dSource, buffer, numPixels * nInputBufferChannels * sizeof(unsigned char),
                                              hipMemcpyHostToDevice));

                        // Allocate device buffer for the first LOD
                        CUDA_CHECK(hipMalloc(&dMipmap, sizeBytes));

                        // Launch fillFirstMipmapKernel
                        int blockSize = 256;
                        int gridSize = (numPixels + blockSize - 1) / blockSize;
                        fillFirstMipmapKernel<<<gridSize, blockSize>>>(
                            dMipmap,
                            dSource,
                            nInputBufferChannels,
                            nChannels,
                            currentSize);
                        CUDA_CHECK(hipDeviceSynchronize());

                        // Copy back to host (mipmapBuffers[0]) for CPU-based compression
                        CUDA_CHECK(hipMemcpy(mipmapBuffers[lod].data(), dMipmap, sizeBytes, hipMemcpyDeviceToHost));

                        // Now you can compress on the CPU side, store to compressed buffer, etc...

                        CUDA_CHECK(hipFree(dSource));
                        CUDA_CHECK(hipFree(dMipmap));
                    }
                    else if (lod < 3)
                    // Next LODs
                    {
                        // Suppose we want to generate mipmapBuffers[lod] from mipmapBuffers[lod - 1]
                        // We can do a similar approach but with fillMipmapKernel.
                        // For each LOD, the "srcSize" = previous LOD dimension, which is 2 * currentSize.
                        // The "dstSize" = currentSize (the new LOD dimension).

                        // 1) Upload mipmapBuffers[lod - 1] to device (dSource).
                        // 2) Allocate dDst for mipmapBuffers[lod].
                        // 3) Launch fillMipmapKernel<<<dimGrid, dimBlock>>>(...).
                        // 4) Copy dDst back to host.

                        int dstSize = currentSize;     // for the new LOD
                        int srcSize = currentSize * 2; // for the old LOD

                        // Example device buffers
                        unsigned char *dSrc = nullptr;
                        unsigned char *dDst = nullptr;

                        size_t srcNumPixels = srcSize * srcSize;
                        size_t dstNumPixels = dstSize * dstSize;

                        CUDA_CHECK(hipMalloc(&dSrc, srcNumPixels * nChannels * sizeof(unsigned char)));
                        CUDA_CHECK(hipMalloc(&dDst, dstNumPixels * nChannels * sizeof(unsigned char)));

                        // Copy previous LOD’s data to dSrc
                        CUDA_CHECK(hipMemcpy(
                            dSrc,
                            mipmapBuffers[lod - 1].data(),
                            srcNumPixels * nChannels * sizeof(unsigned char),
                            hipMemcpyHostToDevice));

                        // Launch kernel
                        dim3 block(16, 16);
                        dim3 grid(
                            (dstSize + block.x - 1) / block.x,
                            (dstSize + block.y - 1) / block.y);

                        fillMipmapKernel<<<grid, block>>>(
                            dDst,
                            dSrc,
                            nChannels,
                            dstSize);
                        CUDA_CHECK(hipDeviceSynchronize());

                        // Copy dDst back to host
                        CUDA_CHECK(hipMemcpy(
                            mipmapBuffers[lod].data(),
                            dDst,
                            dstNumPixels * nChannels * sizeof(unsigned char),
                            hipMemcpyDeviceToHost));

                        // Free device buffers
                        CUDA_CHECK(hipFree(dSrc));
                        CUDA_CHECK(hipFree(dDst));

                        // Continue with compression, etc...
                    }
                    else
                    {
                        for (int x = 0; x < currentSize; ++x)
                        {
                            for (int y = 0; y < currentSize; ++y)
                            {
                                int i = y * currentSize + x;

                                int i00 = (y * 2) * 2 * currentSize + (x * 2);
                                int i01 = (y * 2) * 2 * currentSize + (x * 2 + 1);
                                int i10 = (y * 2 + 1) * 2 * currentSize + (x * 2);
                                int i11 = (y * 2 + 1) * 2 * currentSize + (x * 2 + 1);

                                for (int ch = 0; ch < nChannels; ++ch)
                                {
                                    float val = static_cast<float>(mipmapBuffers[lod - 1][i00 * nChannels + ch]) +
                                                static_cast<float>(mipmapBuffers[lod - 1][i01 * nChannels + ch]) +
                                                static_cast<float>(mipmapBuffers[lod - 1][i10 * nChannels + ch]) +
                                                static_cast<float>(mipmapBuffers[lod - 1][i11 * nChannels + ch]);

                                    val /= 4.0f;
                                    val = std::min(val, 255.0f);

                                    mipmapBuffers[lod][i * nChannels + ch] = static_cast<uint8_t>(val);
                                }
                            }
                        }
                    }

                    stbi_write_png(cacheFileNameDebug.c_str(), currentSize, currentSize, nChannels, mipmapBuffers[lod].data(), currentSize * nChannels);

                    nvtt::RefImage refImage{};
                    refImage.data = mipmapBuffers[lod].data();
                    refImage.width = currentSize;
                    refImage.height = currentSize;
                    refImage.depth = 1;
                    refImage.num_channels = nChannels;

                    nvtt::CPUInputBuffer cpuInputBuffer(&refImage, nvtt::UINT8);

                    int numTiles = cpuInputBuffer.NumTiles();
                    int tileWidth = 0;
                    int tileHeight = 0;
                    cpuInputBuffer.TileSize(tileWidth, tileHeight);

                    int compressedTextureSize = bytesPerTile * numTiles;

                    assert(expectedCompressedTextureSize == compressedTextureSize);
                    assert(tileWidth == 4);
                    assert(tileHeight == 4);

                    if (texture.channel == 4)
                    {
                        nvtt::nvtt_encode_bc7(cpuInputBuffer, true, true, compressedImageBuffer.data(), false, false);
                    }
                    else if (texture.channel == 3)
                    {
                        nvtt::nvtt_encode_bc7(cpuInputBuffer, true, false, compressedImageBuffer.data(), false, false);
                    }
                    else if (texture.channel == 2)
                    {
                        nvtt::nvtt_encode_bc5(cpuInputBuffer, true, compressedImageBuffer.data(), false, false);
                    }
                    else if (texture.channel == 1)
                    {
                        nvtt::nvtt_encode_bc4(cpuInputBuffer, true, compressedImageBuffer.data(), false, false);
                    }
                    else
                    {
                        assert(0);
                    }

                    {
                        std::ofstream myfile(cacheFileName, std::ofstream::out | std::ofstream::binary | std::ofstream::trunc);
                        assert(myfile.is_open());
                        myfile.write(reinterpret_cast<char *>(compressedImageBuffer.data()), compressedTextureSize);
                        myfile.close();
                        std::cout << "Successfully saved cached compressed texture to file \"" << cacheFileName << "\".\n";
                    }
                }

                CUDA_CHECK(hipMemcpy2DToArray(currentMipLevelArray, 0, 0, compressedImageBuffer.data(), pitch, pitch, height, hipMemcpyHostToDevice));
                currentSize >>= 1;
            }

            STBI_FREE(buffer);
        }
    }
}
