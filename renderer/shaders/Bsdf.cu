#include "hip/hip_runtime.h"
#include "SystemParameter.h"
#include "OptixShaderCommon.h"
#include "ShaderDebugUtils.h"

namespace jazzfusion
{

    extern "C" __constant__ SystemParameter sysParam;

    __forceinline__ __device__ void unitSquareToCosineHemisphere(const Float2 sample, Float3 const &axis, Float3 &w, float &pdf)
    {
        // Choose a point on the local hemisphere coordinates about +z.
        const float theta = 2.0f * M_PIf * sample.x;
        const float r = sqrtf(sample.y);
        w.x = r * cosf(theta);
        w.y = r * sinf(theta);
        w.z = 1.0f - w.x * w.x - w.y * w.y;
        w.z = (0.0f < w.z) ? sqrtf(w.z) : 0.0f;

        pdf = w.z * M_1_PIf;

        // Align with axis.
        alignVector(axis, w);
    }

    extern "C" __device__ void __direct_callable__sample_bsdf_diffuse_reflection(MaterialParameter const &parameters, MaterialState const &state, PerRayData *rayData, Float3 &wi, Float3 &f_over_pdf, float &pdf)
    {
        unitSquareToCosineHemisphere(rayData->rand2(sysParam), state.normal, wi, pdf);

        if (!(rayData->flags & FLAG_DIFFUSED))
        {
            rayData->material |= RAY_MAT_FLAG_DIFFUSE << (2 * rayData->depth);
        }

        // if (pdf <= 0.0f || dot(wi, state.normalGeo) <= 0.0f)
        if (pdf <= 0.0f || dot(wi, state.normal) <= 0.0f)
        {
            rayData->flags |= FLAG_TERMINATE;
            return;
        }

        f_over_pdf = Float3(1.0f);
    }

    // The parameter wiL is the lightSample.direction (direct lighting), not the next ray segment's direction rayData.wi (indirect lighting).
    extern "C" __device__ Float4 __direct_callable__eval_bsdf_diffuse_reflection(MaterialParameter const &parameters, MaterialState const &state, PerRayData *const rayData, const Float3 wiL)
    {
        const Float3 f = Float3(1.0f) * M_1_PIf;
        const float pdf = fmaxf(0.0f, dot(wiL, state.normal) * M_1_PIf);

        return Float4(f, pdf);
    }

    __device__ __forceinline__ float pow5(float e)
    {
        float e2 = e * e;
        return e2 * e2 * e;
    }

    __device__ __forceinline__ Float3 FresnelShlick(const Float3 &F0, float cosTheta)
    {
        return F0 + (Float3(1.0f) - F0) * pow5(1.0f - cosTheta);
    }

    __device__ __forceinline__ float FresnelShlick(float F0, float cosTheta)
    {
        return F0 + (1.0f - F0) * pow5(1.0f - cosTheta);
    }

    extern "C" __device__ void __direct_callable__sample_bsdf_microfacet_reflection(MaterialParameter const &parameters, MaterialState const &state, PerRayData *rayData, Float3 &wi, Float3 &brdfOverPdf, float &pdf)
    {
        Float2 r = rayData->rand2(sysParam);

        // Roughness to alpha
        float roughness = state.roughness;
        roughness = max(roughness, 1e-3f);
        float x = logf(roughness);
        float alpha = 1.62142f + 0.819955f * x + 0.1734f * x * x + 0.0171201f * x * x * x + 0.000640711f * x * x * x * x;

        // pre-calculate alpha2
        float alpha2 = alpha * alpha;

        // sample normal
        Float3 sampledNormal;
        float cosTheta = 1.0f / sqrtf(1.0f + alpha2 * r[0] / (1.0f - r[0]));
        float sinTheta = sqrtf(1.0f - cosTheta * cosTheta);
        float phi = TWO_PI * r[1];
        sampledNormal = Float3(sinTheta * cos(phi), sinTheta * sin(phi), cosTheta);

        // local to world space
        alignVector(state.normal, sampledNormal);

        // reflect
        wi = normalize(reflect3f(-state.wo, sampledNormal));

        // if (OPTIX_CENTER_PIXEL())
        // {
        //     OPTIX_DEBUG_PRINT(state.geometricNormal);
        //     OPTIX_DEBUG_PRINT(state.normal);
        //     OPTIX_DEBUG_PRINT(sampledNormal);
        //     OPTIX_DEBUG_PRINT(state.wo);
        //     OPTIX_DEBUG_PRINT(wi);
        // }

        if (dot(wi, state.geometricNormal) < 0)
        {
            Float2 r2 = rayData->rand2(sysParam);

            // Try again
            cosTheta = 1.0f / sqrt(1.0f + alpha2 * r2[0] / (1.0f - r2[0]));
            sinTheta = sqrt(1.0f - cosTheta * cosTheta);
            phi = TWO_PI * r2[1];
            sampledNormal = Float3(sinTheta * cos(phi), sinTheta * sin(phi), cosTheta);
            alignVector(state.normal, sampledNormal);
            wi = normalize(reflect3f(-state.wo, sampledNormal));

            if (dot(wi, state.geometricNormal) < 0)
            {
                // Failed twice
                // wi = normalize(reflect3f(-state.wo, state.geometricNormal));
                brdfOverPdf = Float3(0.0f);
                pdf = 0.0f;
                return;
            }
        }

        Float3 wh = sampledNormal;
        Float3 wn = state.geometricNormal;
        Float3 wo = state.wo;

        // Fresnel (dieletric or metal)
        constexpr float eta1 = 1.4f;
        constexpr float eta2 = 1.0f;
        constexpr float F0 = ((eta1 - eta2) / (eta1 + eta2)) * ((eta1 - eta2) / (eta1 + eta2));
        float cosThetaWoWh = max(SAFE_COSINE_EPSI, dot(wh, wo));
        float F = FresnelShlick(F0, cosThetaWoWh); // Can be Float3

        // Smith's Mask-shadowing function G
        float cosThetaWo = clampf(dot(wo, wn), SAFE_COSINE_EPSI, 1.0f - SAFE_COSINE_EPSI);
        float cosThetaWi = max(SAFE_COSINE_EPSI, dot(wi, wn));
        float tanThetaWo = sqrt(1.0f - cosThetaWo * cosThetaWo) / cosThetaWo;
        float G = 1.0f / (1.0f + (sqrtf(1.0f + alpha2 * tanThetaWo * tanThetaWo) - 1.0f) / 2.0f);

        // Trowbridge Reitz Distribution D
        float cosThetaWh = max(SAFE_COSINE_EPSI, dot(wh, wn));
        float cosTheta2Wh = cosThetaWh * cosThetaWh;
        float tanTheta2Wh = (1.0f - cosTheta2Wh) / cosTheta2Wh;
        float e = tanTheta2Wh / alpha2 + 1.0f;
        float D = 1.0f / (M_PI * (alpha2 * cosTheta2Wh * cosTheta2Wh) * (e * e));

        // brdf
        Float3 brdf = Float3(F) * (D * G) / (4.0f * cosThetaWo * cosThetaWi);

        // pdf
        pdf = (D * cosThetaWh) / (4.0f * cosThetaWoWh);

        // beta
        brdfOverPdf = Float3(F) * (G * cosThetaWoWh) / (cosThetaWh * cosThetaWo); // brdf / pdf * cosThetaWi

        // if (OPTIX_CENTER_PIXEL())
        // {
        //     OPTIX_DEBUG_PRINT(brdf);
        //     OPTIX_DEBUG_PRINT(pdf);
        //     OPTIX_DEBUG_PRINT(brdfOverPdf);
        // }
    }

    extern "C" __device__ Float4 __direct_callable__eval_bsdf_microfacet_reflection(MaterialParameter const &parameters, MaterialState const &state, PerRayData *const rayData, const Float3 wiL)
    {
        Float3 wo = state.wo;
        Float3 wn = state.geometricNormal;
        Float3 wi = wiL;

        Float3 brdf = Float3(0.0f);
        float pdf = 0.0f;

        if (dot(wo, wn) <= 0 || dot(wi, wn) <= 0)
        {
            return Float4(brdf, pdf);
        }

        // Roughness to alpha
        float roughness = state.roughness;
        roughness = max(roughness, 1e-3f);
        float x = logf(roughness);
        float alpha = 1.62142f + 0.819955f * x + 0.1734f * x * x + 0.0171201f * x * x * x + 0.000640711f * x * x * x * x;
        float alpha2 = alpha * alpha;

        Float3 wh = normalize(wi + wo);

        // Fresnel (dieletric or metal)
        constexpr float eta1 = 1.4f;
        constexpr float eta2 = 1.0f;
        constexpr float F0 = ((eta1 - eta2) / (eta1 + eta2)) * ((eta1 - eta2) / (eta1 + eta2));
        float cosThetaWoWh = max(SAFE_COSINE_EPSI, dot(wh, wo));
        float F = FresnelShlick(F0, cosThetaWoWh); // Can be Float3

        // Smith's Mask-shadowing function G
        float cosThetaWo = clampf(dot(wo, wn), SAFE_COSINE_EPSI, 1.0f - SAFE_COSINE_EPSI);
        float cosThetaWi = max(SAFE_COSINE_EPSI, dot(wi, wn));
        float tanThetaWo = sqrtf(1.0f - cosThetaWo * cosThetaWo) / cosThetaWo;
        float G = 1.0f / (1.0f + (sqrtf(1.0f + alpha2 * tanThetaWo * tanThetaWo) - 1.0f) / 2.0f);

        // Trowbridge Reitz Distribution D
        float cosThetaWh = max(SAFE_COSINE_EPSI, dot(wh, wn));
        float cosTheta2Wh = cosThetaWh * cosThetaWh;
        float tanTheta2Wh = (1.0f - cosTheta2Wh) / cosTheta2Wh;
        float e = tanTheta2Wh / alpha2 + 1.0f;
        float D = 1.0f / (M_PI * (alpha2 * cosTheta2Wh * cosTheta2Wh) * (e * e));

        brdf = Float3(F) * (D * G) / (4.0f * cosThetaWo * cosThetaWi);
        pdf = (D * cosThetaWh) / (4.0f * cosThetaWoWh);

        return Float4(brdf, pdf);
    }

    extern "C" __device__ void __direct_callable__sample_bsdf_specular_reflection(MaterialParameter const &parameters, MaterialState const &state, PerRayData *rayData, Float3 &wi, Float3 &f_over_pdf, float &pdf)
    {
        wi = reflect3f(-rayData->wo, state.normal);

        // Do not sample opaque materials below the geometric surface.
        // if (pdf <= 0.0f || dot(wi, state.normalGeo) <= 0.0f)
        if (pdf <= 0.0f || dot(wi, state.normal) <= 0.0f)
        {
            rayData->flags |= FLAG_TERMINATE;
            return;
        }

        if (!(rayData->flags & FLAG_DIFFUSED))
        {
            rayData->material |= RAY_MAT_FLAG_REFL_OR_REFR << (2 * rayData->depth);
        }

        f_over_pdf = Float3(1.0f);
        pdf = 1.0f; // Not 0.0f to make sure the path is not terminated. Otherwise unused for specular events.
    }

    // This function evaluates a Fresnel dielectric function when the transmitting cosine ("cost")
    // is unknown and the incident index of refraction is assumed to be 1.0f.
    // \param et     The transmitted index of refraction.
    // \param costIn The cosine of the angle between the incident direction and normal direction.
    __forceinline__ __device__ float evaluateFresnelDielectric(const float et, const float cosIn)
    {
        const float cosi = fabsf(cosIn);

        float sint = 1.0f - cosi * cosi;
        sint = (0.0f < sint) ? sqrtf(sint) / et : 0.0f;

        // Handle total internal reflection.
        if (1.0f < sint)
        {
            return 1.0f;
        }

        float cost = 1.0f - sint * sint;
        cost = (0.0f < cost) ? sqrtf(cost) : 0.0f;

        const float et_cosi = et * cosi;
        const float et_cost = et * cost;

        const float rPerpendicular = (cosi - et_cost) / (cosi + et_cost);
        const float rParallel = (et_cosi - cost) / (et_cosi + cost);

        const float result = (rParallel * rParallel + rPerpendicular * rPerpendicular) * 0.5f;

        return (result <= 1.0f) ? result : 1.0f;
    }

    extern "C" __device__ void __direct_callable__sample_bsdf_specular_reflection_transmission(MaterialParameter const &parameters, MaterialState const &state, PerRayData *rayData, Float3 &wi, Float3 &f_over_pdf, float &pdf)
    {
        // Return the current material's absorption coefficient and ior to the integrator to be able to support nested materials.
        rayData->absorption_ior = Float4(parameters.absorption, parameters.ior);

        // Need to figure out here which index of refraction to use if the ray is already inside some refractive medium.
        // This needs to happen with the original FLAG_FRONTFACE condition to find out from which side of the geometry we're looking!
        // ior.xy are the current volume's IOR and the surrounding volume's IOR.
        // Thin-walled materials have no volume, always use the frontface eta for them!
        const float eta = (rayData->flags & (FLAG_FRONTFACE | FLAG_THINWALLED))
                              ? rayData->absorption_ior.w / rayData->ior.x
                              : rayData->ior.y / rayData->absorption_ior.w;

        const Float3 R = reflect3f(-rayData->wo, state.normal);

        float reflective = 1.0f;

        if (refract(wi, -rayData->wo, state.normal, eta))
        {
            if (rayData->flags & FLAG_THINWALLED)
            {
                wi = -rayData->wo; // Straight through, no volume.
            }
            // Total internal reflection will leave this reflection probability at 1.0f.
            reflective = evaluateFresnelDielectric(eta, dot(rayData->wo, state.normal));
        }

        if (rayData->flags & FLAG_VOLUME) // If we are inside a volumn
        {
            if (!(rayData->flags & FLAG_DIFFUSED))
            {
                rayData->material |= RAY_MAT_FLAG_REFL_OR_REFR << (2 * rayData->depth);
            }

            if (reflective == 1.0f) // Either total reflection
            {
                wi = R;
            }
            else // Or total transmission
            {
                rayData->flags |= FLAG_TRANSMISSION;
            }
            f_over_pdf = Float3(1.0f);
        }
        else
        {
            if (!(rayData->flags & FLAG_DIFFUSED))
            {
                rayData->material |= RAY_MAT_FLAG_REFR_AND_REFL << (2 * rayData->depth);
            }

            if (rayData->sampleIdx & 0x1)
            {
                wi = R; // Fresnel reflection or total internal reflection.
                f_over_pdf = Float3(reflective);
            }
            else if (!(rayData->flags & FLAG_THINWALLED)) // Only non-thinwalled materials have a volume and transmission events.
            {
                rayData->flags |= FLAG_TRANSMISSION;
                f_over_pdf = Float3(1.0f - reflective);
            }

            // const float pseudo = rayData->rand(sysParam);
            // if (pseudo < reflective)
            // {
            //     wi = R; // Fresnel reflection or total internal reflection.
            // }
            // else if (!(rayData->flags & FLAG_THINWALLED)) // Only non-thinwalled materials have a volume and transmission events.
            // {
            //     rayData->flags |= FLAG_TRANSMISSION;
            // }
        }

        // No Fresnel factor here. The probability to pick one or the other side took care of that.
        // f_over_pdf = Float3(1.0f);
        pdf = 1.0f; // Not 0.0f to make sure the path is not terminated. Otherwise unused for specular events.
    }

}