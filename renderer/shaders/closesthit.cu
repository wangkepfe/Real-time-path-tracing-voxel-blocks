#include "hip/hip_runtime.h"
#include "SystemParameter.h"
#include "OptixShaderCommon.h"
#include "ShaderDebugUtils.h"
#include "Sampler.h"
#include "SelfHit.h"

namespace jazzfusion
{

    extern "C" __constant__ SystemParameter sysParam;

    // Get the 3x4 object to world transform and its inverse from a two-level hierarchy.
    // Arguments Float4* objectToWorld, Float4* worldToObject shortened for smaller code.
    __forceinline__ __device__ void getTransforms(Float4 *mW, Float4 *mO)
    {
        OptixTraversableHandle handle = optixGetTransformListHandle(0);

        const Float4 *tW = reinterpret_cast<const Float4 *>(optixGetInstanceTransformFromHandle(handle));
        const Float4 *tO = reinterpret_cast<const Float4 *>(optixGetInstanceInverseTransformFromHandle(handle));

        mW[0] = tW[0];
        mW[1] = tW[1];
        mW[2] = tW[2];

        mO[0] = tO[0];
        mO[1] = tO[1];
        mO[2] = tO[2];
    }

    // Functions to get the individual transforms in case only one of them is needed.

    __forceinline__ __device__ void getTransformObjectToWorld(Float4 *mW)
    {
        OptixTraversableHandle handle = optixGetTransformListHandle(0);

        const Float4 *tW = reinterpret_cast<const Float4 *>(optixGetInstanceTransformFromHandle(handle));

        mW[0] = tW[0];
        mW[1] = tW[1];
        mW[2] = tW[2];
    }

    __forceinline__ __device__ void getTransformWorldToObject(Float4 *mO)
    {
        OptixTraversableHandle handle = optixGetTransformListHandle(0);

        const Float4 *tO = reinterpret_cast<const Float4 *>(optixGetInstanceInverseTransformFromHandle(handle));

        mO[0] = tO[0];
        mO[1] = tO[1];
        mO[2] = tO[2];
    }

    // Matrix3x4 * point. v.w == 1.0f
    __forceinline__ __device__ Float3 transformPoint(const Float4 *m, Float3 const &v)
    {
        Float3 r;

        r.x = m[0].x * v.x + m[0].y * v.y + m[0].z * v.z + m[0].w;
        r.y = m[1].x * v.x + m[1].y * v.y + m[1].z * v.z + m[1].w;
        r.z = m[2].x * v.x + m[2].y * v.y + m[2].z * v.z + m[2].w;

        return r;
    }

    // Matrix3x4 * vector. v.w == 0.0f
    __forceinline__ __device__ Float3 transformVector(const Float4 *m, Float3 const &v)
    {
        Float3 r;

        r.x = m[0].x * v.x + m[0].y * v.y + m[0].z * v.z;
        r.y = m[1].x * v.x + m[1].y * v.y + m[1].z * v.z;
        r.z = m[2].x * v.x + m[2].y * v.y + m[2].z * v.z;

        return r;
    }

    // InverseMatrix3x4^T * normal. v.w == 0.0f
    // Get the inverse matrix as input and applies it as inverse transpose.
    __forceinline__ __device__ Float3 transformNormal(const Float4 *m, Float3 const &v)
    {
        Float3 r;

        r.x = m[0].x * v.x + m[1].x * v.y + m[2].x * v.z;
        r.y = m[0].y * v.x + m[1].y * v.y + m[2].y * v.z;
        r.z = m[0].z * v.x + m[1].z * v.y + m[2].z * v.z;

        return r;
    }

    // ========================================================================================
    // Example: Overload a small helper to get the safe world-space position (and normal) from a triangle.
    // This example uses the SIA library calls you shared earlier.
    // ========================================================================================
    __forceinline__ __device__ void getSafeTriangleSpawnOffsetInWorldSpace(
        const float3 &v0,
        const float3 &v1,
        const float3 &v2,
        float3 &spawnPos,    // [out] final safe position in world space
        float3 &spawnNormal, // [out] final safe normal in world space
        float &spawnOffset   // [out] final safe offset along normal in world space
    )
    {
        // 3) Get barycentrics for the hit.
        float2 bary = optixGetTriangleBarycentrics();

        // 4) Compute the safe offset *in object space*.
        //    If your GAS is truly in object space, you can do:
        //    SelfIntersectionAvoidance::getSafeTriangleSpawnOffset(objPos, objNorm, objOffset, v0, v1, v2, bary).
        //    However, if your triangle data[] is already in world space, you can skip transformSafeSpawnOffset below.
        //    For demonstration, let's assume data[] is object-space, so we do:
        float3 objPos, objNorm;
        float objOffset;
        SelfIntersectionAvoidance::getSafeTriangleSpawnOffset(
            /*out*/ objPos,
            /*out*/ objNorm,
            /*out*/ objOffset,
            v0, v1, v2, // v0, v1, v2
            bary);

        // 5) Now convert that safe offset into world space.
        //    - If your code uses a single instance transform, the default
        //      transformSafeSpawnOffset(...) will read the local transform list
        //      from optixGetTransformListHandle(0..N).
        //    - If you have multiple levels, you can pass in an array of traversable handles.
        //    - For a single transform, the following call is enough:
        float3 wPos, wNorm;
        float wOffset;
        SelfIntersectionAvoidance::transformSafeSpawnOffset(
            /*out*/ wPos,
            /*out*/ wNorm,
            /*out*/ wOffset,
            objPos, // from step 4
            objNorm,
            objOffset);

        // 6) Provide the final results back to our caller.
        spawnPos = wPos;
        spawnNormal = wNorm;
        spawnOffset = wOffset;
    }

    extern "C" __global__ void __closesthit__radiance()
    {
        PerRayData *rayData = mergePointer(optixGetPayload_0(), optixGetPayload_1());

        rayData->distance = optixGetRayTmax();

        // if (OPTIX_CENTER_PIXEL())
        // {
        //     OPTIX_DEBUG_PRINT(rayData->distance);
        // }

        // rayData->pos = rayData->pos + rayData->wi * rayData->distance;

        GeometryInstanceData *instanceData = reinterpret_cast<GeometryInstanceData *>(optixGetSbtDataPointer());

        const unsigned int thePrimtiveIndex = optixGetPrimitiveIndex();

        const Int3 tri = instanceData->indices[thePrimtiveIndex];

        const VertexAttributes &va0 = instanceData->attributes[tri.x];
        const VertexAttributes &va1 = instanceData->attributes[tri.y];
        const VertexAttributes &va2 = instanceData->attributes[tri.z];

        const Float3 v0 = va0.vertex;
        const Float3 v1 = va1.vertex;
        const Float3 v2 = va2.vertex;

        float2 bary = optixGetTriangleBarycentrics();

        Float3 frontPos;
        Float3 backPos;
        Float3 geometricNormal;
        {
            float3 objPos, objNorm;
            float objOffset;
            SelfIntersectionAvoidance::getSafeTriangleSpawnOffset(
                /*out*/ objPos,
                /*out*/ objNorm,
                /*out*/ objOffset,
                v0.to_float3(), v1.to_float3(), v2.to_float3(), // v0, v1, v2
                bary);

            float3 safePos, safeNorm;
            float safeOffset;
            SelfIntersectionAvoidance::transformSafeSpawnOffset(
                /*out*/ safePos,
                /*out*/ safeNorm,
                /*out*/ safeOffset,
                objPos, // from step 4
                objNorm,
                objOffset);

            float3 tmpFrontPos, tmpBackPos;
            SelfIntersectionAvoidance::offsetSpawnPoint(
                /*out*/ tmpFrontPos,
                /*out*/ tmpBackPos,
                /*position =*/safePos,
                /*direction=*/safeNorm,
                /*offset   =*/safeOffset);

            frontPos = Float3(tmpFrontPos);
            backPos = Float3(tmpBackPos);
            geometricNormal = Float3(safeNorm);
        }

        // Default pos
        rayData->pos = frontPos;

        bool hitFrontFace = dot(rayData->wo, geometricNormal) > 0.0f;

        const MaterialParameter &parameters = sysParam.materialParameters[instanceData->materialIndex];
        int materialId = parameters.indexBSDF;
        rayData->material = (float)materialId;

        if (materialId == INDEX_BSDF_EMISSIVE)
        {
            rayData->radiance = parameters.albedo;
            rayData->shouldTerminate = true;
            return;
        }

        bool isThinfilm = materialId == INDEX_BSDF_DIFFUSE_REFLECTION_TRANSMISSION_THINFILM;

        if (isThinfilm && !hitFrontFace)
        {
            geometricNormal = -geometricNormal;
            hitFrontFace = true;

            Float3 tmp = frontPos;
            frontPos = backPos;
            backPos = tmp;
        }

        if (rayData->isShadowRay)
        {
            rayData->hasShadowRayHitAnything = true;

            if (materialId == INDEX_BSDF_SPECULAR_REFLECTION_TRANSMISSION)
            {
                rayData->hasShadowRayHitTransmissiveSurface = true;
                rayData->absorption_ior.xyz = parameters.absorption;

                if (rayData->isInsideVolume)
                {
                    rayData->pos = frontPos;
                }
                else
                {
                    rayData->pos = backPos;
                }
            }
            else if (isThinfilm)
            {
                rayData->hasShadowRayHitThinfilmSurface = true;
                rayData->absorption_ior.xyz = parameters.absorption;

                rayData->pos = backPos;
            }

            return;
        }

        // UI Box
        if (0)
        {
            if (rayData->depth == 0)
            {
                Float3 highlightPoint[4];
                highlightPoint[0] = sysParam.edgeToHighlight[0];
                highlightPoint[1] = sysParam.edgeToHighlight[1];
                highlightPoint[2] = sysParam.edgeToHighlight[2];
                highlightPoint[3] = sysParam.edgeToHighlight[3];

                const float tolerance = 0.005f;
                Float3 dummy;
                float d0 = PointToSegmentDistance(rayData->pos, highlightPoint[0], highlightPoint[1], dummy);
                float d1 = PointToSegmentDistance(rayData->pos, highlightPoint[1], highlightPoint[2], dummy);
                float d2 = PointToSegmentDistance(rayData->pos, highlightPoint[2], highlightPoint[3], dummy);
                float d3 = PointToSegmentDistance(rayData->pos, highlightPoint[3], highlightPoint[0], dummy);

                if (d0 < tolerance || d1 < tolerance || d2 < tolerance || d3 < tolerance)
                {
                    Store2DFloat4(Float4(1.0f), sysParam.outUiBuffer, Int2(optixGetLaunchIndex().x, optixGetLaunchIndex().y));
                }
            }
        }

        MaterialState state;
        state.geometricNormal = geometricNormal;
        state.wo = rayData->wo;

        const Float2 theBarycentrics = Float2(bary);
        const float alpha = 1.0f - theBarycentrics.x - theBarycentrics.y;

        if (parameters.flags == 2) // use texture coordinates
        {
            state.texcoord = va0.texcoord * alpha + va1.texcoord * theBarycentrics.x + va2.texcoord * theBarycentrics.y;

            // if (OPTIX_CENTER_PIXEL())
            // {
            //     OPTIX_DEBUG_PRINT(state.texcoord);
            //     OPTIX_DEBUG_PRINT(va0.texcoord);
            //     OPTIX_DEBUG_PRINT(va1.texcoord);
            //     OPTIX_DEBUG_PRINT(va2.texcoord);
            //     OPTIX_DEBUG_PRINT(Float3(theBarycentrics.x, theBarycentrics.y, alpha));
            // }
        }
        else
        {
            if (abs(state.geometricNormal.x) > 0.9f)
            {
                state.texcoord.x = fmodf(rayData->pos.z, parameters.uvScale);
                state.texcoord.y = fmodf(rayData->pos.y, parameters.uvScale);
            }
            else if (abs(state.geometricNormal.y) > 0.9f)
            {
                state.texcoord.x = fmodf(rayData->pos.x, parameters.uvScale);
                state.texcoord.y = fmodf(rayData->pos.z, parameters.uvScale);
            }
            else if (abs(state.geometricNormal.z) > 0.9f)
            {
                state.texcoord.x = fmodf(rayData->pos.x, parameters.uvScale);
                state.texcoord.y = fmodf(rayData->pos.y, parameters.uvScale);
            }
        }

        // Ray cone spread
        rayData->rayConeWidth += rayData->rayConeSpread * rayData->distance; // +surfaceRayConeSpread; // @TODO Based on the local surface curvature

        // if (OPTIX_CENTER_PIXEL())
        // {
        //     OPTIX_DEBUG_PRINT(Float4(rayData->pos, rayData->depth));
        //     OPTIX_DEBUG_PRINT(Float4(frontPos, rayData->depth));
        //     OPTIX_DEBUG_PRINT(Float4(backPos, rayData->depth));
        //     OPTIX_DEBUG_PRINT(Float4(rayData->wo, rayData->depth));
        //     OPTIX_DEBUG_PRINT(Float4(state.geometricNormal, rayData->depth));
        // }

        rayData->hitFrontFace = hitFrontFace;

        Float3 albedo = parameters.albedo;

        state.texcoord /= parameters.uvScale;
        float texMip0Size = parameters.texSize.length();
        float lod = log2f(rayData->rayConeWidth / max(dot(state.geometricNormal, rayData->wo), 0.01f) / parameters.uvScale * 2.0f * texMip0Size) - 3.0f;

        if (parameters.textureAlbedo != 0)
        {
            const Float3 texColor = Float3(tex2DLod<float4>(parameters.textureAlbedo, state.texcoord.x, state.texcoord.y, lod));
            albedo *= texColor;

            // if (OPTIX_CENTER_PIXEL())
            // {
            //     OPTIX_DEBUG_PRINT(state.texcoord);
            //     OPTIX_DEBUG_PRINT(texColor);
            //     OPTIX_DEBUG_PRINT(lod);
            // }
        }

        state.albedo = albedo;

        state.roughness = 0.001f;
        if (parameters.textureRoughness != 0)
        {
            state.roughness = tex2DLod<float1>(parameters.textureRoughness, state.texcoord.x, state.texcoord.y, lod).x;
        }

        state.metallic = 0.0f;
        if (parameters.textureMetallic != 0)
        {
            state.metallic = tex2DLod<float1>(parameters.textureMetallic, state.texcoord.x, state.texcoord.y, lod).x;
        }

        if (parameters.textureNormal != 0)
        {
            Float3 texNormal = Float3(tex2DLod<float4>(parameters.textureNormal, state.texcoord.x, state.texcoord.y, lod));
            state.normal = normalize(texNormal - 0.5f);
            state.normal.x = -state.normal.x;
            state.normal.y = -state.normal.y;
            alignVector(state.geometricNormal, state.normal);
        }
        else
        {
            state.normal = state.geometricNormal;
        }

        if (parameters.flags == 1) // water
        {
            if ((abs(state.geometricNormal.x) > 0.9f) || (abs(state.geometricNormal.z) > 0.9f))
            {
                state.normal = state.geometricNormal;
            }
            else
            {
                Float2 texcoord1 = state.texcoord;
                Float2 texcoord2 = state.texcoord;
                texcoord1.x += sysParam.timeInSecond * 0.04f;
                texcoord2 *= 2.0f;
                texcoord2.y += sysParam.timeInSecond * 0.02f;
                Float3 normal1 = Float3(tex2DLod<float4>(parameters.textureNormal, texcoord1.x, texcoord1.y, lod)) - 0.5f;
                Float3 normal2 = Float3(tex2DLod<float4>(parameters.textureNormal, texcoord2.x, texcoord2.y, lod)) - 0.5f;
                state.normal = normalize(normal1 + normal2 * 2.0f);
                alignVector(state.geometricNormal, state.normal);
            }
        }

        rayData->normal = state.normal;
        rayData->roughness = state.roughness;

        bool isDiffuse = materialId >= NUM_SPECULAR_BSDF;

        rayData->isCurrentBounceDiffuse = isDiffuse;

        const int indexBsdfSample = materialId;

        Float3 surfWi;
        Float3 surfBsdfOverPdf;
        float surfSampleSurfPdf;

        optixDirectCall<void, MaterialParameter const &, MaterialState const &, PerRayData *, Float3 &, Float3 &, float &>(indexBsdfSample, parameters, state, rayData, surfWi, surfBsdfOverPdf, surfSampleSurfPdf);

        // if (OPTIX_CENTER_PIXEL())
        // {
        //     OPTIX_DEBUG_PRINT(surfWi);
        //     OPTIX_DEBUG_PRINT(surfBsdfOverPdf);
        //     OPTIX_DEBUG_PRINT(surfSampleSurfPdf);
        // }

        if (isThinfilm)
        {
            rayData->pos = rayData->isHitThinfilmTransmission ? backPos : frontPos;
        }
        else
        {
            if (rayData->hitFrontFace) // front face
            {
                if (rayData->isInsideVolume) // inside volume
                {
                    if (rayData->isHitTransmission) // trasmission
                    {
                        // wrong!
                    }
                    else // reflection
                    {
                        rayData->pos = frontPos;
                    }
                }
                else // outside volumn
                {
                    if (rayData->isHitTransmission) // trasmission
                    {
                        rayData->pos = backPos;
                    }
                    else // reflection
                    {
                        rayData->pos = frontPos;
                    }
                }
            }
            else // backface
            {
                if (rayData->isInsideVolume) // inside volume
                {
                    if (rayData->isHitTransmission) // trasmission
                    {
                        rayData->pos = frontPos;
                    }
                    else // reflection
                    {
                        rayData->pos = backPos;
                    }
                }
                else // outside volumn
                {
                    // pretty wrong situation: pass through

                    if (rayData->isHitTransmission) // trasmission
                    {
                        rayData->pos = backPos;
                    }
                    else // reflection
                    {
                        rayData->pos = frontPos;
                    }

                    rayData->f_over_pdf = Float3(1.0f);
                    rayData->pdf = 1.0f;
                    return;
                }
            }
        }

        if (!rayData->hitFirstDiffuseSurface && isDiffuse)
        {
            rayData->hitFirstDiffuseSurface = true;
            if (rayData->sampleIdx == 0)
            {
                rayData->albedo = albedo;
            }
            else
            {
                rayData->albedo = lerp3f(rayData->albedo, albedo, 1.0f / (float)(rayData->sampleIdx + 1));
            }
            albedo = Float3(1.0f);
        }
        else
        {
            surfBsdfOverPdf *= albedo;
        }

        constexpr bool enableDiffuseOptimization = true;

        if (isDiffuse)
        {
            // Diffuse after diffuse, shadow ray only
            bool shadowRayOnly = false;
            if (enableDiffuseOptimization)
            {
                if (rayData->isLastBounceDiffuse)
                {
                    shadowRayOnly = true;
                }
            }

            // Env light sample
            LightSample lightSample;
            {
                const Int2 skyRes(512, 256);
                const Int2 sunRes(32, 32);

                const float *skyCdf = sysParam.skyCdf;
                const float *sunCdf = sysParam.sunCdf;

                const int skySize = skyRes.x * skyRes.y;
                const int sunSize = sunRes.x * sunRes.y;
                const float sunAngle = 0.51f; // angular diagram in degrees
                const float sunAngleCosThetaMax = cosf(sunAngle * M_PI / 180.0f / 2.0f);

                // The accumulated all sky luminance
                const float maxSkyCdf = skyCdf[skySize - 1];

                // The accumulated all sun luminance
                const float maxSunCdf = sunCdf[sunSize - 1];

                const float totalSkyLum = maxSkyCdf * TWO_PI / skySize; // Jacobian of the hemisphere mapping
                const float totalSunLum = maxSunCdf * TWO_PI * (1.0f - sunAngleCosThetaMax) / sunSize;

                // Sample sky or sun pdf
                const float sampleSkyVsSun = totalSkyLum / (totalSkyLum + totalSunLum);

                if (sampleSkyVsSun > rayData->rand(sysParam))
                {
                    // Binary search in range 0 to size-2, since we want result+1 to be the index, we'll need to subtract result for calculating PDF
                    const int sampledSkyIdx = BinarySearch(skyCdf, 0, skySize - 2, rayData->rand(sysParam) * maxSkyCdf) + 1;

                    // Subtract neighbor CDF to get PDF, divided by max CDF to get the probability
                    float sampledSkyPdf = (skyCdf[sampledSkyIdx] - skyCdf[sampledSkyIdx - 1]) / maxSkyCdf;

                    // Each tile has area 2Pi / resolution, pdf = 1/area = resolution / 2Pi
                    sampledSkyPdf = sampledSkyPdf * skySize / TWO_PI;

                    // Index to 2D coordinates
                    Int2 skyIdx(sampledSkyIdx % skyRes.x, sampledSkyIdx / skyRes.x);
                    float u = (skyIdx.x + 0.5f) / skyRes.x;
                    float v = (skyIdx.y + 0.5f) / skyRes.y;

                    // Hemisphere projection
                    Float3 rayDir = EqualAreaMap(u, v);

                    // Load sky buffer
                    Float3 skyEmission = Load2DFloat4(sysParam.skyBuffer, skyIdx).xyz;

                    // Set light sample direction and PDF
                    lightSample.direction = rayDir;
                    lightSample.pdf = sampledSkyPdf;
                    lightSample.emission = skyEmission;
                }
                else // Choose to sample sun
                {
                    // Binary search in range 0 to size-2, since we want result+1 to be the index, we'll need to subtract result for calculating PDF
                    const int sampledSunIdx = BinarySearch(sunCdf, 0, sunSize - 2, rayData->rand(sysParam) * maxSunCdf) + 1;

                    // Subtract neighbor CDF to get PDF, divided by max CDF to get the probability
                    float sampledSunPdf = (sunCdf[sampledSunIdx] - sunCdf[sampledSunIdx - 1]) / maxSunCdf;

                    // Each tile has area = coneAnglularArea / resolution, pdf = 1/area = resolution / (TWO_PI * (1.0f - cosThetaMax))
                    sampledSunPdf = sampledSunPdf * sunSize / (TWO_PI * (1.0f - sunAngleCosThetaMax));

                    // Index to 2D coordinates
                    Int2 sunIdx(sampledSunIdx % sunRes.x, sampledSunIdx / sunRes.x);
                    float u = (sunIdx.x + 0.5f) / sunRes.x;
                    float v = (sunIdx.y + 0.5f) / sunRes.y;

                    // Hemisphere projection
                    Float3 rayDir = EqualAreaMapCone(sysParam.sunDir, u, v, sunAngleCosThetaMax);

                    // Load sky buffer
                    Float3 sunEmission = Load2DFloat4(sysParam.sunBuffer, sunIdx).xyz;

                    // Set light sample direction and PDF
                    lightSample.direction = rayDir;
                    lightSample.pdf = sampledSunPdf;
                    lightSample.emission = sunEmission;
                }
            }

            float lightSampleLightDistPdf = lightSample.pdf;

            bool isLightGeometricallyVisible;

            if (isThinfilm)
            {
                isLightGeometricallyVisible = true;
            }
            else
            {
                isLightGeometricallyVisible = dot(lightSample.direction, state.geometricNormal) > 0.0f;
            }

            if (0.0f < lightSampleLightDistPdf && isLightGeometricallyVisible) // Valid light sample, verify light distribution
            {
                const int indexBsdfEval = indexBsdfSample + 1;
                const Float4 lightSampleSurfDistBsdfPdf = optixDirectCall<Float4, MaterialParameter const &, MaterialState const &, PerRayData const *, const Float3>(indexBsdfEval, parameters, state, rayData, lightSample.direction);
                Float3 lightSampleSurfDistBsdf = lightSampleSurfDistBsdfPdf.xyz;
                float lightSampleSurfDistPdf = lightSampleSurfDistBsdfPdf.w;

                if (0.0f < lightSampleSurfDistPdf) // Valid light sample, verify surface distribution
                {
                    rayData->isShadowRay = true;

                    Float3 originalPos = rayData->pos;
                    float originalDistance = rayData->distance;
                    Float3 originalAbsorption = rayData->absorption_ior.xyz;
                    bool originalIsInsideVolume = rayData->isInsideVolume;

                    Float3 glassThroughput = Float3(1.0f);

                    if (isThinfilm)
                    {
                        bool visibleFromFrontFace = dot(lightSample.direction, state.geometricNormal) > 0.0f;
                        rayData->pos = visibleFromFrontFace ? frontPos : backPos;

                        const float cosTheta = abs(dot(lightSample.direction, state.normal));
                        glassThroughput = visibleFromFrontFace ? Float3(1.0f) : (rayData->absorption_ior.xyz * cosTheta / M_PI / 0.75f);
                    }

                    rayData->wi = lightSample.direction;

                    // For glass, go *straight* through the surface and volumn to calculate the direct light contribution
                    // Very wrong caustics but cheap enough and look decent
                    for (int shadowRayIter = 0; shadowRayIter < 5; ++shadowRayIter)
                    {
                        UInt2 payload = splitPointer(rayData);

                        rayData->hasShadowRayHitAnything = false;
                        rayData->hasShadowRayHitTransmissiveSurface = false;
                        rayData->hasShadowRayHitThinfilmSurface = false;

                        optixTrace(sysParam.topObject,
                                   (float3)rayData->pos, (float3)rayData->wi,
                                   sysParam.sceneEpsilon, RayMax, 0.0f, // tmin, tmax, time
                                   OptixVisibilityMask(0xFF), OPTIX_RAY_FLAG_DISABLE_ANYHIT,
                                   0, 1, 0,
                                   payload.x, payload.y);

                        if (!rayData->hasShadowRayHitAnything)
                        {
                            float misWeightLightSample = powerHeuristic(lightSampleLightDistPdf, lightSampleSurfDistPdf);
                            const float cosTheta = fmaxf(0.0f, dot(lightSample.direction, state.normal));
                            Float3 shadowRayBsdfOverPdf = lightSampleSurfDistBsdf * cosTheta / lightSampleLightDistPdf;
                            Float3 shadowRayRadiance = lightSample.emission * misWeightLightSample * shadowRayBsdfOverPdf * glassThroughput * albedo;
                            rayData->radiance += shadowRayRadiance;
                            break;
                        }
                        else
                        {
                            if (rayData->hasShadowRayHitTransmissiveSurface)
                            {
                                if (rayData->isInsideVolume)
                                {
                                    rayData->isInsideVolume = false;
                                    glassThroughput *= exp3f(-rayData->distance * rayData->absorption_ior.xyz);
                                }
                                else
                                {
                                    rayData->isInsideVolume = true;
                                }
                                continue;
                            }
                            else if (rayData->hasShadowRayHitThinfilmSurface)
                            {
                                const float cosTheta = abs(dot(lightSample.direction, state.normal));
                                glassThroughput *= rayData->absorption_ior.xyz * cosTheta / M_PI / 0.75f;
                                continue;
                            }
                            else
                            {
                                break;
                            }
                        }
                    }

                    rayData->pos = originalPos;
                    rayData->distance = originalDistance;
                    rayData->absorption_ior.xyz = originalAbsorption;
                    rayData->isInsideVolume = originalIsInsideVolume;

                    rayData->isShadowRay = false;

                    if (enableDiffuseOptimization)
                    {
                        if (shadowRayOnly)
                        {
                            rayData->shouldTerminate = true;
                            return;
                        }
                    }
                }
            }
        }

        rayData->wi = surfWi;
        rayData->f_over_pdf = surfBsdfOverPdf;
        rayData->pdf = surfSampleSurfPdf;
    }
}