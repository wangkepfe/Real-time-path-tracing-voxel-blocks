#include "hip/hip_runtime.h"
#include "OptixShaderCommon.h"
#include "SystemParameter.h"
#include "ShaderDebugUtils.h"
#include "Sampler.h"
#include "Restir.h"

extern "C" __constant__ SystemParameter sysParam;

extern "C" __global__ void __miss__radiance()
{
    RayData *rayData = (RayData *)mergePointer(optixGetPayload_0(), optixGetPayload_1());

    Int2 pixelPosition = Int2(optixGetLaunchIndex());

    Float3 emission = Float3(0);

    const bool enableRIS = true && rayData->depth == 0;
    const bool enableReSTIR = true && enableRIS;
    if (enableReSTIR)
    {
        StoreDIReservoir(EmptyDIReservoir(), pixelPosition);
    }

    if (rayData->depth == 0)
    {
        Store2DFloat4(Float4(1.0f), sysParam.albedoBuffer, pixelPosition);
        Store2DFloat1((float)(0xFFFF), sysParam.materialBuffer, pixelPosition);
        Store2DFloat4(Float4(0.0f, -1.0f, 0.0f, 0.0f), sysParam.normalRoughnessBuffer, pixelPosition);
        Store2DFloat4(Float4(0.0f, -1.0f, 0.0f, 0.0f), sysParam.geoNormalThinfilmBuffer, pixelPosition);
        Store2DFloat4(Float4(0.0f, 0.0f, 0.0f, 0.0f), sysParam.materialParameterBuffer, pixelPosition);
    }

    rayData->lastMissWasEnvironment = true;

    // if (rayData->isLastBounceDiffuse)
    // {
    //     rayData->distance = RayMax;
    //     rayData->shouldTerminate = true;
    //     return;
    // }

    const Int2 &skyRes = sysParam.skyRes;
    const Int2 &sunRes = sysParam.sunRes;
    const int skySize = skyRes.x * skyRes.y;
    const float &accumulatedSkyLuminance = sysParam.accumulatedSkyLuminance;
    const float sunAngle = 0.51f; // angular diagram in degrees
    const float sunAngleCosThetaMax = cosf(sunAngle * M_PI / 180.0f / 2.0f);

    const Float3 &rayDir = rayData->wi;

    // Map the ray diretcion to uv
    Float2 uv;

    // Sky
    {
        uv = EqualAreaSphereMap(rayDir);
        Float3 skyEmission = SampleBicubicSmoothStep<Load2DFuncFloat4<Float3>, Float3, BoundaryFuncRepeatXClampY>(sysParam.skyBuffer, uv, skyRes);
        emission += skyEmission;
    }

    // Is the ray hitting the solar disk, then compute the sun emission
    bool hitDisk = EqualAreaMapCone(uv, sysParam.sunDir, rayDir, sunAngleCosThetaMax);
    if (hitDisk)
    {
        Int2 sunIdx((int)(uv.x * sunRes.x), (int)(uv.y * sunRes.y));

        // Wrapping around on X dimension
        if (sunIdx.x >= sunRes.x)
        {
            sunIdx.x %= sunRes.x;
        }
        if (sunIdx.x < 0)
        {
            sunIdx.x = sunRes.x - (-sunIdx.x) % sunRes.x;
        }
        Float3 sunEmission = Load2DFloat4(sysParam.sunBuffer, sunIdx).xyz;
        emission += sunEmission;
    }

    rayData->radiance = emission;
    rayData->distance = RayMax;
    rayData->shouldTerminate = true;
}

extern "C" __global__ void __miss__bsdf_light()
{
    ShadowRayData *rayData = (ShadowRayData *)mergePointer(optixGetPayload_0(), optixGetPayload_1());
    rayData->lightIdx = SkyLightIndex;
}

extern "C" __global__ void __miss__visibility()
{
    bool *isVisible = (bool *)mergePointer(optixGetPayload_0(), optixGetPayload_1());
    *isVisible = true;
}

