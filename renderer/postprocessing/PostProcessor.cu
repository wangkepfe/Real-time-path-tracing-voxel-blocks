#include "hip/hip_runtime.h"
#include "postprocessing/PostProcessor.h"
#include "postprocessing/ScalingFilter.h"
#include "postprocessing/BicubicFilter.h"
#include "postprocessing/Tonemapping.h"
#include "postprocessing/SharpeningFilter.h"
#include "core/BufferManager.h"
#include "core/GlobalSettings.h"
#include "util/KernelHelper.h"
#include "util/DebugUtils.h"

namespace jazzfusion
{

    __global__ void CopyToInteropBuffer(
        Float4 *out,
        SurfObj outColorBuffer,
        Int2 outSize)
    {
        Int2 idx;
        idx.x = blockIdx.x * blockDim.x + threadIdx.x;
        idx.y = blockIdx.y * blockDim.y + threadIdx.y;

        if (idx.x >= outSize.x || idx.y >= outSize.y)
            return;
        int linearId = idx.y * outSize.x + idx.x;

        Float3 color = Load2DFloat4(outColorBuffer, idx).xyz;

        // if (CUDA_CENTER_PIXEL())
        // {
        //     DEBUG_PRINT(color);
        // }

        out[linearId] = Float4(color, 0);
    }

    void PostProcessor::run(Float4 *interopBuffer, int inputWidthIn, int inputHeightIn, int outputWidthIn, int outputHeightIn)
    {
        inputWidth = inputWidthIn;
        inputHeight = inputHeightIn;
        outputWidth = outputWidthIn;
        outputHeight = outputHeightIn;

        auto &bufferManager = BufferManager::Get();
        const auto &postProcessParams = GlobalSettings::GetPostProcessParams();
        const auto &renderPassSettings = GlobalSettings::GetRenderPassSettings();

        // ToneMappingReinhardExtended KERNEL_ARGS2(GetGridDim(inputWidth, inputHeight, BLOCK_DIM_8x8x1), GetBlockDim(BLOCK_DIM_8x8x1))(bufferManager.GetBuffer2D(RenderColorBuffer), Int2(inputWidth, inputHeight), postProcessParams);

        // if (renderPassSettings.enableSharpening)
        // {
        //     SharpeningFilter KERNEL_ARGS2(GetGridDim(outputWidth, outputHeight, BLOCK_DIM_8x8x1), GetBlockDim(BLOCK_DIM_8x8x1))(bufferManager.GetBuffer2D(RenderColorBuffer), Int2(inputWidth, inputHeight));
        // }

        // if (renderPassSettings.enableEASU)
        // {
        //     EdgeAdaptiveSpatialUpsampling KERNEL_ARGS2(GetGridDim(outputWidth, outputHeight, BLOCK_DIM_8x8x1), GetBlockDim(BLOCK_DIM_8x8x1))(bufferManager.GetBuffer2D(OutputColorBuffer), bufferManager.GetBuffer2D(RenderColorBuffer),
        //                                                                                                                                      inputWidth, inputHeight, inputWidth, inputHeight, outputWidth, outputHeight);
        // }
        // else
        // {
        BicubicFilter KERNEL_ARGS2(GetGridDim(outputWidth, outputHeight, BLOCK_DIM_8x8x1), GetBlockDim(BLOCK_DIM_8x8x1))(bufferManager.GetBuffer2D(OutputColorBuffer), bufferManager.GetBuffer2D(RenderColorBuffer), Int2(inputWidth, inputHeight), Int2(outputWidth, outputHeight));
        // }

        CopyToInteropBuffer KERNEL_ARGS2(GetGridDim(outputWidth, outputHeight, BLOCK_DIM_8x8x1), GetBlockDim(BLOCK_DIM_8x8x1))(interopBuffer, bufferManager.GetBuffer2D(OutputColorBuffer), Int2(outputWidth, outputHeight));

        CUDA_CHECK(hipDeviceSynchronize());
        CUDA_CHECK(hipPeekAtLastError());
    }

}