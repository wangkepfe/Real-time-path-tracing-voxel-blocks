#include "hip/hip_runtime.h"
#include "postprocessing/PostProcessingPipeline.h"
#include "util/KernelHelper.h"
#include "util/DebugUtils.h"
#include "core/BufferManager.h"
#include "core/RenderCamera.h"
#include "sky/Sky.h"
#include "shaders/Common.h"
#include "shaders/Sampler.h"
#include <hip/hip_runtime.h>

// Bloom bright pass extraction kernel with neighbor filtering
__global__ void BloomExtractBrightPixelsKernel(
    SurfObj inputBuffer,
    SurfObj outputBuffer,
    Int2 size,
    float threshold,
    bool useNeighborFilter)
{
    Int2 idx;
    idx.x = blockIdx.x * blockDim.x + threadIdx.x;
    idx.y = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx.x >= size.x || idx.y >= size.y)
        return;

    Float3 color = Load2DFloat4(inputBuffer, idx).xyz;

    // Compute luminance
    float luminance = dot(color, Float3(0.2126f, 0.7152f, 0.0722f));

    // Extract bright pixels above threshold
    if (luminance > threshold)
    {
        bool isValidBloomPixel = true;

        // Neighbor filtering to reduce fireflies in bloom
        if (useNeighborFilter)
        {
            float maxNeighborLum = 0.0f;

            // Check immediate neighbors (+ pattern)
            Int2 neighbors[4] = {
                Int2(idx.x - 1, idx.y), Int2(idx.x + 1, idx.y),
                Int2(idx.x, idx.y - 1), Int2(idx.x, idx.y + 1)};

            for (int i = 0; i < 4; i++)
            {
                if (neighbors[i].x >= 0 && neighbors[i].x < size.x &&
                    neighbors[i].y >= 0 && neighbors[i].y < size.y)
                {
                    Float3 neighborColor = Load2DFloat4(inputBuffer, neighbors[i]).xyz;
                    float neighborLum = dot(neighborColor, Float3(0.2126f, 0.7152f, 0.0722f));
                    maxNeighborLum = fmaxf(maxNeighborLum, neighborLum);
                }
            }

            // Require at least one neighbor to be somewhat bright
            if (maxNeighborLum < threshold * 0.4f)
            {
                isValidBloomPixel = false;
            }
        }

        if (isValidBloomPixel)
        {
            // Preserve color while scaling down by threshold (reduced intensity)
            color = clamp3f((color - Float3(threshold)) * 0.7f, Float3(0.0f), Float3(100.0f));
        }
        else
        {
            color = Float3(0.0f);
        }
    }
    else
    {
        color = Float3(0.0f);
    }

    Store2DFloat4(Float4(color, 1.0f), outputBuffer, idx);
}

// Gaussian blur kernel for bloom
__global__ void BloomBlurKernel(
    SurfObj inputBuffer,
    SurfObj outputBuffer,
    Int2 size,
    Int2 direction, // (1,0) for horizontal, (0,1) for vertical
    float radius)
{
    Int2 idx;
    idx.x = blockIdx.x * blockDim.x + threadIdx.x;
    idx.y = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx.x >= size.x || idx.y >= size.y)
        return;

    Float3 result = Float3(0.0f);
    float totalWeight = 0.0f;

    // Simple box blur with radius
    int kernelSize = (int)(radius * 2.0f) + 1;
    int halfKernel = kernelSize / 2;

    for (int i = -halfKernel; i <= halfKernel; i++)
    {
        Int2 samplePos = idx + direction * i;

        // Clamp to buffer bounds
        samplePos.x = clampf(samplePos.x, 0, size.x - 1);
        samplePos.y = clampf(samplePos.y, 0, size.y - 1);

        Float3 sample = Load2DFloat4(inputBuffer, samplePos).xyz;
        float weight = 1.0f; // Simple box filter

        result += sample * weight;
        totalWeight += weight;
    }

    if (totalWeight > 0.0f)
    {
        result /= totalWeight;
    }

    Store2DFloat4(Float4(result, 1.0f), outputBuffer, idx);
}

// Bloom composite kernel
__global__ void BloomCompositeKernel(
    SurfObj colorBuffer, // Original scene color (modified in-place)
    SurfObj bloomBuffer, // Blurred bloom texture
    Int2 size,
    float intensity)
{
    Int2 idx;
    idx.x = blockIdx.x * blockDim.x + threadIdx.x;
    idx.y = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx.x >= size.x || idx.y >= size.y)
        return;

    Float3 originalColor = Load2DFloat4(colorBuffer, idx).xyz;
    Float3 bloomColor = Load2DFloat4(bloomBuffer, idx).xyz;

    // Additive blending with intensity control
    Float3 finalColor = originalColor + bloomColor * intensity;

    Store2DFloat4(Float4(finalColor, 1.0f), colorBuffer, idx);
}

// Vignette effect kernel
__global__ void VignetteKernel(
    SurfObj colorBuffer,
    Int2 size,
    float strength,
    float radius,
    float smoothness)
{
    Int2 idx;
    idx.x = blockIdx.x * blockDim.x + threadIdx.x;
    idx.y = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx.x >= size.x || idx.y >= size.y)
        return;

    Float3 color = Load2DFloat4(colorBuffer, idx).xyz;

    // Calculate normalized coordinates from center (-1 to 1)
    float x = (2.0f * idx.x - size.x) / (float)size.x;
    float y = (2.0f * idx.y - size.y) / (float)size.y;

    // Calculate distance from center
    float distance = sqrtf(x * x + y * y);

    // Apply vignette falloff using manual smoothstep
    float t = clampf((distance - radius) / smoothness, 0.0f, 1.0f);
    float smoothed = t * t * (3.0f - 2.0f * t); // smoothstep interpolation
    float vignette = 1.0f - smoothed;
    vignette = 1.0f - strength * (1.0f - vignette);
    vignette = clampf(vignette, 0.0f, 1.0f);

    // Apply vignette to color
    color *= vignette;

    Store2DFloat4(Float4(color, 1.0f), colorBuffer, idx);
}

static bool ProjectSunToScreen(const Camera &camera, const Float3 &sunDir, Int2 size, Float2 &outUV, Int2 &outPixel)
{
    Float3 uvw = camera.worldToUv * sunDir;
    if (uvw.z <= 0.0f)
        return false;

    Float2 uv = Float2(uvw.x, uvw.y) / uvw.z;
    if (uv.x < 0.0f || uv.x > 1.0f || uv.y < 0.0f || uv.y > 1.0f)
        return false;

    auto clampTo = [](int value, int minValue, int maxValue) {
        return value < minValue ? minValue : (value > maxValue ? maxValue : value);
    };

    outUV = uv;
    outPixel = Int2(
        clampTo(static_cast<int>(uv.x * size.x), 0, size.x - 1),
        clampTo(static_cast<int>(uv.y * size.y), 0, size.y - 1));
    return true;
}

static bool IsSunVisible(const Int2 &sunPixel, hipArray_t depthArray)
{
    float depthValue = RayMax;
    CUDA_CHECK(hipMemcpy2DFromArray(
        &depthValue,
        sizeof(float),
        depthArray,
        static_cast<size_t>(sunPixel.x) * sizeof(float),
        static_cast<size_t>(sunPixel.y),
        sizeof(float),
        1,
        hipMemcpyDeviceToHost));
    return depthValue >= RayMaxLowerBound;
}

__global__ void LensFlareKernel(
    SurfObj colorBuffer,
    Int2 size,
    Float2 sunUV,
    float sunLuminance,
    float intensity,
    float ghostSpacing,
    int ghostCount,
    float haloRadius,
    float sunSize,
    float distortion)
{
    Int2 idx;
    idx.x = blockIdx.x * blockDim.x + threadIdx.x;
    idx.y = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx.x >= size.x || idx.y >= size.y)
        return;

    Float3 originalColor = Load2DFloat4(colorBuffer, idx).xyz;
    Float3 flareColor = Float3(0.0f);

    Float2 uv = Float2((float)idx.x / size.x, (float)idx.y / size.y);
    Float2 center = Float2(0.5f, 0.5f);

    float aspect = (float)size.x / (float)size.y;
    Float2 uvAspect = Float2(uv.x * aspect, uv.y);
    Float2 sunUVAspect = Float2(sunUV.x * aspect, sunUV.y);
    Float2 centerAspect = Float2(center.x * aspect, center.y);

    Float2 toSunAspect = uvAspect - sunUVAspect;
    float dist = sqrtf(toSunAspect.x * toSunAspect.x + toSunAspect.y * toSunAspect.y);

    Float2 sunToCenterAspect = centerAspect - sunUVAspect;
    float axisDistance = sqrtf(sunToCenterAspect.x * sunToCenterAspect.x + sunToCenterAspect.y * sunToCenterAspect.y);

    float effectiveSunSize = fmaxf(sunSize, 0.0005f);
    float lightIntensity = fmaxf(sunLuminance, 1.0f);

    if (axisDistance > 0.0001f)
    {
        Float2 axisDir = sunToCenterAspect / axisDistance;

        if (dist < effectiveSunSize)
        {
            float sunFalloff = 1.0f - (dist / effectiveSunSize);
            sunFalloff = sunFalloff * sunFalloff;
            flareColor += Float3(1.0f, 0.9f, 0.7f) * sunFalloff * intensity * lightIntensity * 0.1f;
        }

        if (haloRadius > 0.0001f)
        {
            float haloFalloff = expf(-dist * dist / (haloRadius * haloRadius));
            flareColor += Float3(1.0f, 0.8f, 0.6f) * haloFalloff * intensity * lightIntensity * 0.08f;
        }

        for (int g = 1; g <= ghostCount; ++g)
        {
            float ghostFactor = ghostSpacing * static_cast<float>(g);
            float ghostDistance = fminf(ghostFactor, 1.0f) * axisDistance;
            Float2 ghostCenterAspect = sunUVAspect + axisDir * ghostDistance;

            Float2 toGhostAspect = uvAspect - ghostCenterAspect;
            float ghostDist = sqrtf(toGhostAspect.x * toGhostAspect.x + toGhostAspect.y * toGhostAspect.y);

            float ghostSize = 0.02f + (g % 3) * 0.01f;
            float ghostFalloff = expf(-ghostDist * ghostDist / (ghostSize * ghostSize));

            Float3 ghostTint;
            switch (g % 4)
            {
            case 0: ghostTint = Float3(1.0f, 0.7f, 0.3f); break;
            case 1: ghostTint = Float3(0.8f, 1.0f, 0.5f); break;
            case 2: ghostTint = Float3(0.6f, 0.8f, 1.0f); break;
            default: ghostTint = Float3(1.0f, 0.6f, 0.8f); break;
            }

            float ghostIntensity = intensity * lightIntensity * 0.04f * (1.0f - (float)g / fmaxf((float)ghostCount, 1.0f));
            flareColor += ghostTint * ghostFalloff * ghostIntensity;
        }

        if (distortion > 0.0f)
        {
            float aberrationStart = fmaxf(effectiveSunSize * 1.5f, 0.02f);
            float aberrationFade = clampf((dist - aberrationStart) / 0.5f, 0.0f, 1.0f);
            float aberrationStrength = distortion * intensity * lightIntensity * 0.02f;
            float falloff = (1.0f / (1.0f + dist * 6.0f)) * aberrationFade * aberrationFade;
            flareColor += Float3(aberrationStrength, 0.0f, -aberrationStrength) * falloff;
        }
    }

    Float3 finalColor = originalColor + flareColor;
    Store2DFloat4(Float4(finalColor, 1.0f), colorBuffer, idx);
}

// Simple luminance histogram computation kernel
__global__ void ComputeLuminanceHistogramKernel(
    SurfObj colorBuffer,
    Int2 size,
    float *histogram,
    int numBins,
    float minLogLum,
    float maxLogLum)
{
    Int2 idx;
    idx.x = blockIdx.x * blockDim.x + threadIdx.x;
    idx.y = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx.x >= size.x || idx.y >= size.y)
        return;

    Float3 color = Load2DFloat4(colorBuffer, idx).xyz;

    // Compute luminance
    float luminance = dot(color, Float3(0.2126f, 0.7152f, 0.0722f));

    // Skip dark pixels
    if (luminance < 0.001f)
        return;

    // Map to histogram bin
    float logLum = log10f(luminance);
    float t = clampf((logLum - minLogLum) / (maxLogLum - minLogLum), 0.0f, 1.0f);
    int bin = min((int)(t * numBins), numBins - 1);

    // Atomic add to histogram
    atomicAdd(&histogram[bin], 1.0f);
}

// Compute average luminance from histogram
__global__ void ComputeAverageLuminanceKernel(
    float *histogram,
    int numBins,
    float minLogLum,
    float maxLogLum,
    float histogramMinPercent,
    float histogramMaxPercent,
    float *avgLuminance)
{
    if (threadIdx.x != 0 || blockIdx.x != 0)
        return;

    // Sum total pixels
    float totalPixels = 0.0f;
    for (int i = 0; i < numBins; i++)
    {
        totalPixels += histogram[i];
    }

    if (totalPixels == 0.0f)
    {
        *avgLuminance = 0.18f; // Default middle gray
        return;
    }

    // Find range based on percentiles
    float minCount = totalPixels * histogramMinPercent / 100.0f;
    float maxCount = totalPixels * histogramMaxPercent / 100.0f;

    float accumCount = 0.0f;
    int minBin = 0, maxBin = numBins - 1;

    // Find min bin
    for (int i = 0; i < numBins; i++)
    {
        accumCount += histogram[i];
        if (accumCount >= minCount)
        {
            minBin = i;
            break;
        }
    }

    // Find max bin
    accumCount = 0.0f;
    for (int i = 0; i < numBins; i++)
    {
        accumCount += histogram[i];
        if (accumCount >= maxCount)
        {
            maxBin = i;
            break;
        }
    }

    // Compute average in range
    float weightedSum = 0.0f;
    float weightTotal = 0.0f;

    for (int i = minBin; i <= maxBin; i++)
    {
        float binCenter = minLogLum + (i + 0.5f) * (maxLogLum - minLogLum) / numBins;
        weightedSum += histogram[i] * binCenter;
        weightTotal += histogram[i];
    }

    if (weightTotal > 0.0f)
    {
        float avgLogLum = weightedSum / weightTotal;
        *avgLuminance = powf(10.0f, avgLogLum);
    }
    else
    {
        *avgLuminance = 0.18f;
    }
}

PostProcessingPipeline::PostProcessingPipeline()
    : d_luminanceHistogram(nullptr)
    , histogramBins(256)
    , m_currentAvgLuminance(0.18f)
    , m_targetAvgLuminance(0.18f)
    , m_lastFrameTime(0.0f)
{
}

PostProcessingPipeline::~PostProcessingPipeline()
{
    if (d_luminanceHistogram)
    {
        hipFree(d_luminanceHistogram);
        d_luminanceHistogram = nullptr;
    }
}

void PostProcessingPipeline::Initialize(int width, int height)
{
    // Allocate histogram buffer
    if (!d_luminanceHistogram)
    {
        hipMalloc(&d_luminanceHistogram, histogramBins * sizeof(float));
    }
}

float PostProcessingPipeline::Execute(
    SurfObj colorBuffer,
    Int2 size,
    const PostProcessingPipelineParams &pipelineParams,
    const ToneMappingParams &toneMappingParams,
    float deltaTime)
{
    float computedExposure = toneMappingParams.manualExposure;

    // Auto-exposure calculation
    if (pipelineParams.enableAutoExposure)
    {
        // Clear histogram
        hipMemset(d_luminanceHistogram, 0, histogramBins * sizeof(float));

        // Compute luminance histogram
        ComputeLuminanceHistogramKernel<<<GetGridDim(size.x, size.y, BLOCK_DIM_8x8x1), GetBlockDim(BLOCK_DIM_8x8x1)>>>(
            colorBuffer,
            size,
            d_luminanceHistogram,
            histogramBins,
            -8.0f, // minLogLum
            4.0f   // maxLogLum
        );

        // Compute average luminance
        float *d_avgLum;
        hipMalloc(&d_avgLum, sizeof(float));

        ComputeAverageLuminanceKernel<<<1, 1>>>(
            d_luminanceHistogram,
            histogramBins,
            -8.0f, // minLogLum
            4.0f,  // maxLogLum
            pipelineParams.histogramMinPercent,
            pipelineParams.histogramMaxPercent,
            d_avgLum);

        float avgLum;
        hipMemcpy(&avgLum, d_avgLum, sizeof(float), hipMemcpyDeviceToHost);
        hipFree(d_avgLum);

        // Smooth adaptation
        m_targetAvgLuminance = avgLum;
        float adaptSpeed = pipelineParams.exposureSpeed * deltaTime;
        m_currentAvgLuminance = lerp(m_currentAvgLuminance, m_targetAvgLuminance, clampf(adaptSpeed, 0.0f, 1.0f));

        // Compute exposure from average luminance
        float keyValue = pipelineParams.targetLuminance;
        computedExposure = keyValue / max(m_currentAvgLuminance, 0.001f);

        // Apply exposure compensation
        computedExposure *= powf(2.0f, pipelineParams.exposureCompensation);

        // Clamp to min/max exposure
        float minExposure = powf(2.0f, pipelineParams.exposureMin);
        float maxExposure = powf(2.0f, pipelineParams.exposureMax);
        computedExposure = clampf(computedExposure, minExposure, maxExposure);
    }

    // Bloom effect
    if (pipelineParams.enableBloom)
    {
        auto &bufferManager = BufferManager::Get();

        // Extract bright pixels
        BloomExtractBrightPixelsKernel<<<GetGridDim(size.x, size.y, BLOCK_DIM_8x8x1), GetBlockDim(BLOCK_DIM_8x8x1)>>>(
            colorBuffer,
            bufferManager.GetBuffer2D(BloomExtractBuffer),
            size,
            pipelineParams.bloomThreshold,
            true);

        // Horizontal blur pass
        BloomBlurKernel<<<GetGridDim(size.x, size.y, BLOCK_DIM_8x8x1), GetBlockDim(BLOCK_DIM_8x8x1)>>>(
            bufferManager.GetBuffer2D(BloomExtractBuffer),
            bufferManager.GetBuffer2D(BloomTempBuffer),
            size,
            Int2(1, 0), // Horizontal direction
            pipelineParams.bloomRadius);

        // Vertical blur pass
        BloomBlurKernel<<<GetGridDim(size.x, size.y, BLOCK_DIM_8x8x1), GetBlockDim(BLOCK_DIM_8x8x1)>>>(
            bufferManager.GetBuffer2D(BloomTempBuffer),
            bufferManager.GetBuffer2D(BloomExtractBuffer),
            size,
            Int2(0, 1), // Vertical direction
            pipelineParams.bloomRadius);

        // Composite bloom back onto original image
        BloomCompositeKernel<<<GetGridDim(size.x, size.y, BLOCK_DIM_8x8x1), GetBlockDim(BLOCK_DIM_8x8x1)>>>(
            colorBuffer,
            bufferManager.GetBuffer2D(BloomExtractBuffer),
            size,
            pipelineParams.bloomIntensity);
    }

    // Lens flare effect (before vignette so it can be darkened by vignette)
    if (pipelineParams.enableLensFlare)
    {
        auto &bufferManager = BufferManager::Get();
        const Camera &camera = RenderCamera::Get().camera;
        const auto &skyModel = SkyModel::Get();
        Float3 sunDir = normalize(skyModel.getSunDir());

        Float2 sunUV;
        Int2 sunPixel;
        if (ProjectSunToScreen(camera, sunDir, size, sunUV, sunPixel))
        {
            hipArray_t depthArray = bufferManager.GetBufferArray(DepthBuffer);
            if (IsSunVisible(sunPixel, depthArray))
            {
                float sunLuminance = skyModel.getAccumulatedSunLuminance();
                if (!(sunLuminance > 0.0f))
                    sunLuminance = 1.0f;

                LensFlareKernel<<<GetGridDim(size.x, size.y, BLOCK_DIM_8x8x1), GetBlockDim(BLOCK_DIM_8x8x1)>>>(
                    colorBuffer,
                    size,
                    sunUV,
                    sunLuminance,
                    pipelineParams.lensFlareIntensity,
                    pipelineParams.lensFlareGhostSpacing,
                    pipelineParams.lensFlareGhostCount,
                    pipelineParams.lensFlareHaloRadius,
                    pipelineParams.lensFlareSunSize,
                    pipelineParams.lensFlareDistortion);
                CUDA_CHECK(hipGetLastError());
            }
        }
    }

    // Vignette effect
    if (pipelineParams.enableVignette)
    {
        VignetteKernel<<<GetGridDim(size.x, size.y, BLOCK_DIM_8x8x1), GetBlockDim(BLOCK_DIM_8x8x1)>>>(
            colorBuffer,
            size,
            pipelineParams.vignetteStrength,
            pipelineParams.vignetteRadius,
            pipelineParams.vignetteSmoothness);
    }

    return computedExposure;
}
